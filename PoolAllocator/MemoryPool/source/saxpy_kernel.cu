
#include <hip/hip_runtime.h>

__host__ __device__ inline void saxpy_kernel( int i, float a, float const* __restrict__ const x, float * __restrict__ const y ){
    y[i] = a * x[i] + y[i];
}

void saxpy_cpu( int n, float a, float const* __restrict__ const x, float * __restrict__ const y ){
    for ( int i = 0; i < n; ++i )
        saxpy_kernel(i,a,x,y);
}

__global__ void saxpy_gpu( int n, float a, float const* __restrict__ const x, float * __restrict__ const y ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n ) return;
    saxpy_kernel(i,a,x,y);
}
