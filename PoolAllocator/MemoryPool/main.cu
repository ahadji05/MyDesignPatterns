#include "hip/hip_runtime.h"

#include "Dispatching.hpp"
#include "SaxpyKernel.hpp"

#include <chrono>
#include <vector>

const size_t POOL_SIZE = 1E+9;

int main() {
try
{
    HostMemoryPool pool( POOL_SIZE, 4096 );

    const auto start { std::chrono::steady_clock::now() };
    int k = 0;
    while ( k < 1 ) {
        char * ptr = ( char* ) pool.allocate( 100000 );
        if ( !ptr ) break;

        float a = 1.5;
        float *x = ( float* ) ptr;
        float *y = ( float* ) ( ptr + 50000 );

        // zero-copy dispatch of SaxpyKernel based on the type of the pool
        dispatch_from( pool, SaxpyKernel{}, 12500, a, x, y );

        auto ptr1 = pool.allocate( 4096 );
        if ( !ptr1 ) break;

        auto ptr2 = pool.allocate( 4097 );
        if ( !ptr2 ) break;

        pool.deallocate( ptr2 );
        pool.deallocate( ptr1 );
        pool.deallocate( ptr );

        ++k;
        if ( k % 100000 == 0 ) std::cout << k << std::endl;
    }
    std::cout << "max k : " <<  k << std::endl;
    const auto finish { std::chrono::steady_clock::now() };
    const std::chrono::duration<double> elapsed_seconds{ finish - start };
    std::cout << "time: " << elapsed_seconds.count() << " seconds\n";
}
catch(const std::exception& e)
{
    std::cerr << e.what() << '\n';
}
    return EXIT_SUCCESS;
}
