#include "hip/hip_runtime.h"

#include "CudaMemoryPool.hpp"
#include "ContiguousMemoryPool.hpp"
#include "AlignedMemoryPool.hpp"
#include <chrono>

const size_t POOL_SIZE = 1E+9;

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

int main() {

    CudaMemoryPool pool( POOL_SIZE, 4096 );

    const auto start { std::chrono::steady_clock::now() };
    int i = 0;
    while ( i < 5 ) {
        char * ptr = ( char* ) pool.allocate( 100000 );
        if ( !ptr ) break;

        float *a = ( float* ) ptr;
        float *b = ( float* ) ( ptr + 50000 );
        // for ( int j = 0; j < 12500; ++j )
        //     a[j] += b[j];

        saxpy<<<99,1024>>>(100000, 1, a, b);

        auto ptr1 = pool.allocate( 4096 );
        if ( !ptr1 ) break;

        auto ptr2 = pool.allocate( 4097 );
        if ( !ptr2 ) break;

        pool.deallocate( ptr2 );
        pool.deallocate( ptr1 );
        pool.deallocate( ptr );

        ++i;
        if ( i % 100000 == 0 ) std::cout << i << std::endl;
    }
    std::cout << "max i : " <<  i << std::endl;
    const auto finish { std::chrono::steady_clock::now() };
    const std::chrono::duration<double> elapsed_seconds{ finish - start };
    std::cout << "time: " << elapsed_seconds.count() << " seconds\n";

    return EXIT_SUCCESS;
}
